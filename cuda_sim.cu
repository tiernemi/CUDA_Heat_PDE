#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  cuda_sim.cu
 *
 *    Description:  Source file for Heat flow simulation for radiator class on the GPU.
 *
 *        Version:  1.0
 *        Created:  19/03/16 17:48:31
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Michael Tierney (MT), tiernemi@tcd.ie
 *
 * =====================================================================================
 */

#include "cuda_sim.hpp"
#include "stdio.h"

// Two alternating surface buffers to eliminate expensive copying. //
surface<void, 2> gpuSurfBuf1 ; // Surface Buffer 
surface<void, 2> gpuSurfBuf2 ; // Surface Buffer


/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  simulateRowSegment21(
 *    Arguments:  int pipeLength - Length of individual pipe.
 *                int numPipes - Number of pipes.
 *  Description:  Performs nearest neighbour approximations of pipe with periodic
 *                boundaries. Reads from buffer 2 and writes to buffer 1. Shared memory
 *                reduces the cost of accessing neighbours.
 * =====================================================================================
 */

template <typename T>
__global__ void simulateRowSegment21(int pipeLength, int numPipes) {

	int threadID = threadIdx.x + blockDim.x*blockIdx.x ;	
	int rowID = blockIdx.y*blockDim.y + threadIdx.y ; 
	extern __shared__ T oldRowData[] ;
	int elementID = threadIdx.x + 2 ;
	int globalID = threadID + 2 ;
	int rowOffset = 0 ;
	if (blockIdx.x == (pipeLength-2)/blockDim.x) {
		rowOffset = threadIdx.y*(pipeLength-blockDim.x*blockIdx.x+2) ;
	} else {
		rowOffset = threadIdx.y*(blockDim.x+4) ;
	}
	
	if (globalID < pipeLength && rowID < numPipes) {
		if (threadIdx.x == 0) {
			surf2Dread(&oldRowData[rowOffset+0], gpuSurfBuf1, (threadID)*sizeof(T), rowID) ;
			surf2Dread(&oldRowData[rowOffset+1], gpuSurfBuf1, (threadID+1)*sizeof(T), rowID) ;
		}
		if (threadIdx.x == blockDim.x-1 || globalID == pipeLength-1) {
			surf2Dread(&oldRowData[rowOffset+elementID+1], gpuSurfBuf1, (globalID+1)*sizeof(T), rowID) ; 
			surf2Dread(&oldRowData[rowOffset+elementID+2], gpuSurfBuf1, (globalID+2)*sizeof(T), rowID) ; 
		}
		surf2Dread(&oldRowData[rowOffset+elementID], gpuSurfBuf1, (globalID)*sizeof(T), rowID) ; 
		__syncthreads() ;

		T newVal =  (0.37*oldRowData[rowOffset+elementID-2]) + (0.28*oldRowData[rowOffset+elementID-1]) + 0.2*oldRowData[rowOffset+elementID] + 
			(0.12*oldRowData[rowOffset+elementID+1]) + (0.03*oldRowData[rowOffset+elementID+2]) ;

		surf2Dwrite(newVal, gpuSurfBuf2, globalID*sizeof(T), rowID); 
	} else {
		__syncthreads() ;
	}
}
template __global__ void simulateRowSegment21<float>(int, int) ;

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  simulateRowSegment12(
 *    Arguments:  int pipeLength - Length of individual pipe.
 *                int numPipes - Number of pipes.
 *  Description:  Performs nearest neighbour approximations of pipe with periodic
 *                boundaries. Reads from buffer 1 and writes to buffer 2. Shared memory
 *                reduces the cost of accessing neighbours.
 * =====================================================================================
 */

template <typename T>
__global__ void simulateRowSegment12(int pipeLength, int numPipes) {

	int threadID = threadIdx.x + blockDim.x*blockIdx.x ;	
	int rowID = blockIdx.y*blockDim.y + threadIdx.y ; 
	extern __shared__ T oldRowData[] ;
	int elementID = threadIdx.x + 2 ;
	int globalID = threadID + 2 ;
	int rowOffset = 0 ;
	if (blockIdx.x == (pipeLength-2)/blockDim.x) {
		rowOffset = threadIdx.y*(pipeLength-blockDim.x*blockIdx.x+2) ;
	} else {
		rowOffset = threadIdx.y*(blockDim.x+4) ;
	}

	if (globalID < pipeLength && rowID < numPipes) {
		if (threadIdx.x == 0) {
			surf2Dread(&oldRowData[rowOffset+0], gpuSurfBuf2, (threadID)*sizeof(T), rowID) ;
			surf2Dread(&oldRowData[rowOffset+1], gpuSurfBuf2, (threadID+1)*sizeof(T), rowID) ;
		}
		if (threadIdx.x == blockDim.x-1 || globalID == pipeLength-1) {
			surf2Dread(&oldRowData[rowOffset+elementID+1], gpuSurfBuf2, (globalID+1)*sizeof(T), rowID) ; 
			surf2Dread(&oldRowData[rowOffset+elementID+2], gpuSurfBuf2, (globalID+2)*sizeof(T), rowID) ; 
		}
		surf2Dread(&oldRowData[rowOffset+elementID], gpuSurfBuf2, (globalID)*sizeof(T), rowID) ; 
		__syncthreads() ;

		T newVal = (0.37*oldRowData[rowOffset+elementID-2]) + (0.28*oldRowData[rowOffset+elementID-1]) + 0.2*oldRowData[rowOffset+elementID] + 
			(0.12*oldRowData[rowOffset+elementID+1]) + (0.03*oldRowData[rowOffset+elementID+2]) ;

		surf2Dwrite(newVal, gpuSurfBuf1, globalID*sizeof(T), rowID); 
	} else {
		__syncthreads() ;
	}
}
template __global__ void simulateRowSegment12<float>(int, int) ;

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  copy_kernel
 *    Arguments:  int width - Width of array to be copied.
 *                int height - Height of array to be copied.
 *  Description:  Copies surface data from one surface to another.
 * =====================================================================================
 */

template <typename T>
__global__ void copyKernel(int width, int height) { 
	// Calculate surface coordinates 
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; 
	if (x < width && y < height) { 
		T data; 
		// Read from input surface 
		surf2Dread(&data, gpuSurfBuf2, x * sizeof(T), y); 
		// Write to output surface 
		surf2Dwrite(data, gpuSurfBuf1, x * sizeof(T), y); 
	} 
}
template __global__ void copyKernel<float>(int, int) ;


/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  transformTextureToGlobal
 *    Arguments:  T * gpu_odata - Pointer to global array on GPU.
 *                int width - Width of array.
 *                int height - Height of array.
 *  Description:  Copies surface to global memory.
 * =====================================================================================
 */

template <typename T>
__global__ void transformTextureToGlobal (T * gpu_odata, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
				
	if ( (x < width) && (y < height) ) {
		surf2Dread(&(gpu_odata[y*width+x]), gpuSurfBuf1, x*sizeof(T) , y); 
	}
}
template __global__ void transformTextureToGlobal<float>(float *, int, int) ;


/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  cudaSimulateHeatFlow
 *    Arguments:  int numIters - Number of iterations of the simulation.
 *                int numPipes - Number of independent pipes in simulation.
 *                int pipeLength - Number of simulation sites for each pipe.
 *                std::vector<float> & times - Vector storing timing data.
 *                int numThreadPerBlockX - Number of threads in x dimension of 2D block.
 *                int numThreadPerBlockY - Number of threads in y dimension of 2D block.
 *  Description:  Copies simulation initial conditions to GPU, simulates each pipe over
 *                a 2D grid using shared and surface memory and then copies the resulting
 *                data back to the CPU.
 * =====================================================================================
 */

template <typename T>
void cudaSimulateHeatFlow(int numIters, int numPipes, int pipeLength, T * data, std::vector<float> & times, 
		int numThreadPerBlockX, int numThreadPerBlockY) {

	int totSize = numPipes*(pipeLength+2) ;
	// Allocation Timing. //
	float elapsedTime ;
	hipEvent_t start, finish ;
	hipEventCreate(&start) ;
	hipEventCreate(&finish) ;
	hipEventRecord(start, 0) ;

	// Allocate memory for Array. //
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>() ;
	hipArray * bufferArray1 ; 
	hipArray * bufferArray2 ; 
	T * gpuDataGlobal ;
	hipMalloc( (void**) &gpuDataGlobal, totSize*sizeof(T)) ;
	hipMallocArray(&bufferArray1, &channelDesc, pipeLength+2, numPipes, hipArraySurfaceLoadStore) ; 
	hipMallocArray(&bufferArray2, &channelDesc, pipeLength+2 , numPipes, hipArraySurfaceLoadStore) ; 
	
	hipEventRecord(finish, 0) ;
	hipEventSynchronize(finish) ;
	// Timing for transfer CPU->GPU. //
	hipEventElapsedTime(&elapsedTime, start, finish);
	times.push_back(elapsedTime) ;

	hipEventRecord(start, 0) ;
    hipMemcpyToArray(bufferArray2, 0, 0, data, totSize*sizeof(T), hipMemcpyHostToDevice) ; 
    hipMemcpyToArray(bufferArray1, 0, 0, data, totSize*sizeof(T), hipMemcpyHostToDevice) ; 
	cudaBindSurfaceToArray(gpuSurfBuf2, bufferArray1) ; 
	cudaBindSurfaceToArray(gpuSurfBuf1, bufferArray2) ; 

	dim3 cpDimBlock(32, 32, 1);
	dim3 cpDimGrid((pipeLength+2)/cpDimBlock.x + ((!(pipeLength+2)%cpDimBlock.x)?0:1), (numPipes)/ cpDimBlock.y + (!(numPipes%cpDimBlock.y)?0:1), 1);
	dim3 dimBlock(numThreadPerBlockX, numThreadPerBlockY, 1) ;
	dim3 dimGrid(pipeLength/dimBlock.x + (!(pipeLength%dimBlock.x)?0:1), (numPipes)/dimBlock.y + (!(numPipes%dimBlock.y)?0:1)) ;

	hipEventRecord(finish, 0) ;
	hipEventSynchronize(finish) ;
	// Timing for Compute on GPU . //
	hipEventElapsedTime(&elapsedTime, start, finish);
	times.push_back(elapsedTime) ;
	hipEventRecord(start, 0) ;

	
	for (int j = 0 ; j < numIters/2 ; ++j) {
		simulateRowSegment21<T><<<dimGrid,dimBlock,(numThreadPerBlockX+4)*(numThreadPerBlockY)*sizeof(T)>>>(pipeLength, numPipes) ;
		simulateRowSegment12<T><<<dimGrid,dimBlock,(numThreadPerBlockX+4)*(numThreadPerBlockY)*sizeof(T)>>>(pipeLength, numPipes) ;	
	} if (numIters % 2 == 1) {
		simulateRowSegment21<T><<<dimGrid,dimBlock,(numThreadPerBlockX+4)*(numThreadPerBlockY)*sizeof(T)>>>(pipeLength, numPipes) ;
		copyKernel<T><<<cpDimGrid, cpDimBlock>>>(pipeLength+2, numPipes) ;
	}

	hipEventRecord(finish, 0) ;
	hipEventSynchronize(finish) ;
	// Timing for transfer GPU -> CPU //
	hipEventElapsedTime(&elapsedTime, start, finish);
	times.push_back(elapsedTime) ;
	hipEventRecord(start, 0) ;

	transformTextureToGlobal<T><<<cpDimGrid, cpDimBlock>>>(gpuDataGlobal,pipeLength+2,numPipes) ;
	hipError_t err = hipGetLastError() ;

	hipEventRecord(finish, 0) ;
	hipEventSynchronize(finish) ;
	hipEventElapsedTime(&elapsedTime, start, finish);
	times.push_back(elapsedTime) ;

	float totTime = 0.f ;
	for (int i = 0 ; i < times.size() ; ++i) {
		totTime += times[i] ;
	}
	times.push_back(totTime) ;

	hipMemcpy(data, gpuDataGlobal, totSize*sizeof(T), hipMemcpyDeviceToHost) ;

    hipFreeArray(bufferArray2); 
	hipFreeArray(bufferArray1); 
	hipFree(gpuDataGlobal) ;
	hipDeviceReset() ;
}

// Explicit instantiation of float function. //
template void cudaSimulateHeatFlow<float>(int numIters, int numPipes, int pipeLength, float * data, std::vector<float> & times, 
		int numThreadPerBlockX, int numThreadPerBlock) ;
